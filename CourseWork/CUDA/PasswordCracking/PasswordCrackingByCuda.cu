#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/*********************************************************************************************
//To Compile:

// nvcc -o PasswordCrackingByCuda PasswordCrackingByCuda.cu 

// To Run: ./PasswordCrackingByCuda

*****************************************************************************************/
__device__ int is_a_match(char *check) { // compares each password check

//Initilizing passwords
	char password1[]="AN9810";
	char password2[]="JI2205";
	char password3[]="TM5298";
	char password4[]="UN6085";

 
	char *c1 = check;
	char *c2 = check;
	char *c3 = check;
	char *c4 = check;

	char *pw1 = password1;
	char *pw2 = password2;
	char *pw3 = password3;
	char *pw4 = password4;

	
	while(*c1 == *pw1){

		if(*c1 == '\0'){
		
			return 1;		
		}
		c1++;
		pw1++;
	}


	while(*c2 == *pw2){
		
		if(*c2 == '\0'){
			return 1;
		}
		c2++;
		pw2++;

	}


	while(*c3 == *pw3){
		
		if(*c3 == '\0'){
			return 1;
		}
		c3++;
		pw3++;

	}


	while(*c4 == *pw4){
		
		if(*c4 == '\0'){
			return 1;
		}
		c4++;
		pw4++;

	}

   return 0; // returns 0

}


__global__ void kernel() {

	char check[7];
	check[6] = '\0';
	char i, j, k, l;
	int m = blockIdx.x+65;
	int n = threadIdx.x+65;
	char firstValue = m;
	char secondValue = n;

	check[0] = firstValue;
	check[1] = secondValue;
	for(i='0';i<='9';i++){
		for(j='0'; j<='9'; j++){
			for(k='0'; k<='9'; k++){
				for(l='0'; l<='9'; l++){
				  check[2] = i;
				  check[3] = j;
				  check[4] = k;
				  check[5] = l;

				  if(is_a_match(check)){
					printf("Password successfully cracked: %s\n", check);
				  }
				  //else {
				       //printf("tried: %s\n", check);
				  //}
				}
			}
		}
	 }
}


int time_difference(struct timespec *start, struct timespec *finish, long long int *difference)
 {
	  long long int ds =  finish->tv_sec - start->tv_sec; 
	  long long int dn =  finish->tv_nsec - start->tv_nsec; 

	  if(dn < 0 ) 
	  {
	    ds--;
	    dn += 1000000000; 
          } 

	  *difference = ds * 1000000000 + dn;
	  return !(*difference > 0);
}

//main
int main(int argc, char *argv[])
{
	struct timespec start, finish;   
  	long long int time_elapsed;

  	clock_gettime(CLOCK_MONOTONIC, &start);

 	kernel <<<26, 26>>>();

  	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &finish);
        time_difference(&start, &finish, &time_elapsed);
	  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,(time_elapsed/1.0e9)); 
  return 0;
}





