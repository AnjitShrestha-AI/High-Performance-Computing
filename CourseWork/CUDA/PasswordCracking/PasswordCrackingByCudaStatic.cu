#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>


// nvcc -o PasswordCrackingByCudaStatic PasswordCrackingByCudaStatic.cu 

__device__ int is_a_match(char *check) {
	char password1[]="AN9810";
	char password2[]="JI2205";
	char password3[]="TM5298";
	char password4[]="UN6085";

 
	char *c1 = check;
	char *c2 = check;
	char *c3 = check;
	char *c4 = check;

	char *pw1 = password1;
	char *pw2 = password2;
	char *pw3 = password3;
	char *pw4 = password4;

	
	while(*c1 == *pw1){

		if(*c1 == '\0'){
		
			return 1;		
		}
		c1++;
		pw1++;
	}


	while(*c2 == *pw2){
		
		if(*c2 == '\0'){
			return 1;
		}
		c2++;
		pw2++;

	}


	while(*c3 == *pw3){
		
		if(*c3 == '\0'){
			return 1;
		}
		c3++;
		pw3++;

	}


	while(*c4 == *pw4){
		
		if(*c4 == '\0'){
			return 1;
		}
		c4++;
		pw4++;

	}

   return 0;

}


__global__ void kernel() {

 //char *alphabet = "ABCDEFGHIJKLMNOPQRSTUVWXYZ";
  char alpha[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
  //alphabet[27] = '\0';

  char num[10] = {'0','1','2','3','4','5','6','7','8','9'};
  //numbers[11] = '\0';

  char check[7];
  check[6] = '\0';
  int i, j, k, l;

	for(i=0;i<10;i++){
		for(j=0; j<10; j++){
			for(k=0; k<10; k++){
				for(l=0; l<10; l++){
					
				  check[0] = alpha[blockIdx.x];
				  check[1] = alpha[threadIdx.x];
				  check[2] = num[i];
				  check[3] = num[j];
				  check[4] = num[k];
				  check[5] = num[l];

				  if(is_a_match(check)){
					printf("Password successfully cracked: %s\n", check);
				  }
				  //else {
				       //printf("tried: %s\n", check);
				  //}
				}
			}
		}
	 }
}

int time_difference(struct timespec *start, struct timespec *finish, long long int *difference)
 {
	  long long int ds =  finish->tv_sec - start->tv_sec; 
	  long long int dn =  finish->tv_nsec - start->tv_nsec; 

	  if(dn < 0 ) 
	  {
	    ds--;
	    dn += 1000000000; 
          } 

	  *difference = ds * 1000000000 + dn;
	  return !(*difference > 0);
}





	//Calculating time



int main(int argc, char *argv[])
{

	
 

  	
	struct timespec start, finish;   
  	long long int time_elapsed;

  	clock_gettime(CLOCK_MONOTONIC, &start);
 kernel <<<26, 26>>>();

  hipDeviceSynchronize();


	clock_gettime(CLOCK_MONOTONIC, &finish);
	  time_difference(&start, &finish, &time_elapsed);
	  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
		                                 (time_elapsed/1.0e9)); 
  return 0;
}





